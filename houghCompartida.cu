#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include "common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int gloID = blockID * blockDim.x + threadID;

    // Definir locID usando los IDs de los hilos del bloque
    int locID = threadIdx.x;

    // Definir un acumulador local en memoria compartida llamado localAcc
    extern __shared__ int localAcc[];
   
    // Inicializar a 0 todos los elementos de este acumulador local
    for (int i = locID; i < degreeBins * rBins; i += blockDim.x) {
        localAcc[i] = 0;
    }

    // Barrera para asegurar que todos los hilos hayan completado la inicialización del acumulador local
    __syncthreads();

    if (gloID < w * h) { // Reemplazar el return por una condición para continuar solo si el hilo está dentro de la imagen
        int xCent = w / 2;
        int yCent = h / 2;
        int xCoord = gloID % w - xCent;
        int yCoord = yCent - gloID / w;

        if (pic[gloID] > 0) {
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
                float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
                int rIdx = (r + rMax) / rScale;

                // Coordinar el acceso a memoria y garantizar que la operación de suma sea completada por cada hilo
                atomicAdd(&localAcc[rIdx * degreeBins + tIdx], 1);
            }
        }
    }

    // Barrera para asegurar que todos los hilos hayan completado el proceso de incremento del acumulador local
    __syncthreads();

    // Agregar un loop para sumar los valores del acumulador local localAcc al acumulador global acc
    for (int i = locID; i < degreeBins * rBins; i += blockDim.x) {
        atomicAdd(&acc[i], localAcc[i]);
    }
}


// Se calculan puntos de la línea inicial y final
double getPoints(double val, char op, double angulo) {
    if (op == '+') {
        return val + 1000 * (angulo);
    } else if (op == '-') {
        return val - 1000 * (angulo);
    } else {
        return 0.0; // Valor predeterminado si el operador no es válido
    }
}

// Función para dibujar las líneas más pesadas en la imagen
void drawAllLines(cv::Mat& image, int *h_hough, int w, int h, float rScale, float rMax, int threshold) {
    std::vector<std::pair<cv::Vec2f, int>> linesWithWeights; // Vector para almacenar las líneas con su peso

    for (int r = 0; r < rBins; r++) {
        for (int theta = 0; theta < degreeBins; theta++) {
            int index = r * degreeBins + theta;
            int weight = h_hough[index];
            if (weight > threshold) { // Comprobar si el peso es mayor que el umbral
                float rValue = (r * rScale) - (rMax);
                float thetaValue = theta * radInc;
                linesWithWeights.push_back(std::make_pair(cv::Vec2f(thetaValue, rValue), weight));
            }
        }
    }

    // Ordenar las líneas por peso en orden descendente
    std::sort(linesWithWeights.begin(), linesWithWeights.end(), [](const std::pair<cv::Vec2f, int>& point0, const std::pair<cv::Vec2f, int>& point1) { return point0.second > point1.second;});

    for (int i = 0; i < linesWithWeights.size(); ++i) {
        cv::Vec2f lineParams = linesWithWeights[i].first;
        float theta = lineParams[0], r = lineParams[1];
        double cosTheta = cos(theta), sinTheta = sin(theta);
        double x0 = (w / 2) + (r * cosTheta), y0 = (h / 2) - (r * sinTheta);
        double xA = getPoints(x0, '+', sinTheta), xB = getPoints(x0, '-', sinTheta), 
        yA = getPoints(y0, '+', cosTheta), yB = getPoints(y0, '-', cosTheta);

        cv::line(image, cv::Point(cvRound(xA), cvRound(yA)), cv::Point(cvRound(xB), cvRound(yB)), cv::Scalar(0, 255, 255), 1.75, cv::LINE_AA);
    }

    cv::imwrite("output.png", image);
}


// Función para comparar los resultados y registrar discrepancias
bool compareResults(int* gpuResult, int* cpuResult, int size) {
    bool match = true;
    for (int i = 0; i < size; i++) {
        if (gpuResult[i] != cpuResult[i]) {
            match = false;
            printf("Discrepancia en el índice %d: GPU = %d, CPU = %d\n", i, gpuResult[i], cpuResult[i]);
        }
    }
    return match;
}


//*****************************************************************
int main(int argc, char **argv) {
    if (argc != 2) {
        std::cerr << "Uso: " << argv[0] << " <nombre_de_imagen.pgm>" << std::endl;
        return -1;
    }

    // Load the image using OpenCV
    cv::Mat originalImage = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);

    if (originalImage.empty()) {
        std::cerr << "Error al cargar la imagen." << std::endl;
        return -1;
    }

    int w = originalImage.cols;
    int h = originalImage.rows;

    float *d_Cos;
    float *d_Sin;

    // CPU calculation
    int *cpuResult;
    CPU_HoughTran(originalImage.data, originalImage.cols, originalImage.rows, &cpuResult);

    hipMalloc((void **)&d_Cos, sizeof(float) * degreeBins);
    hipMalloc((void **)&d_Sin, sizeof(float) * degreeBins);

    // pre-compute values to be stored
    float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
    float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (int i = 0; i < degreeBins; i++) {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    hipMemcpy(d_Cos, pcCos, sizeof(float) * degreeBins, hipMemcpyHostToDevice);
    hipMemcpy(d_Sin, pcSin, sizeof(float) * degreeBins, hipMemcpyHostToDevice);

    // setup and copy data from host to device
    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;

    h_in = originalImage.data; // h_in contiene los pixeles de la imagen
    h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);
    
    // Marcar el inicio del tiempo de ejecución del kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
    //Configurar la dimensión del bloque y memoria compartida
    dim3 blockSize(256);
    dim3 gridSize((w * h + blockSize.x - 1) / blockSize.x);
    int sharedMemorySize = degreeBins * rBins * sizeof(int);

    // Llamada al kernel con memoria compartida
    GPU_HoughTran<<<gridSize, blockSize, sharedMemorySize>>>(d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

    // get results from device
    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    // compare CPU and GPU results
    bool resultsMatch = compareResults(h_hough, cpuResult, degreeBins * rBins);

    if (resultsMatch) {
        printf("Los resultados coinciden entre GPU y CPU.\n");
    } else {
        printf("Los resultados difieren entre GPU y CPU.\n");
    }

    // Crea una copia de la imagen original utilizando OpenCV
    cv::Mat imageWithLines;
    cv::cvtColor(originalImage, imageWithLines, cv::COLOR_GRAY2BGR); // Convierte a imagen en color

    int threshold = 4175; // Define la cantidad máxima de líneas a dibujar
    drawAllLines(imageWithLines, h_hough, w, h, rScale, rMax, threshold);

    // Marcar el final del tiempo de ejecución del kernel
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Tiempo de ejecución del kernel: %f ms\n", milliseconds);

    printf("Done!\n");

    free(pcCos);
    free(pcSin);
    free(h_hough);
    free(cpuResult);
    hipFree(d_Cos);
    hipFree(d_Sin);
    hipFree(d_in);
    hipFree(d_hough);

    return 0;
}